#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <algorithm>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "graph.h"
#include "parse_graph.hpp"
#include "limits.h"

/****** START UTIL METHODS ******/
bool edgeSrcComparator(edge a, edge b){ return (a.src < b.src); }
bool edgeDestComparator(edge a, edge b){ return (a.dest < b.dest); }

void swap(void **a, void **b){
    void *tmp = *a;
    *a = *b;
    *b = tmp;
}

int readCudaInt(int *i){
    int tmp;
    hipMemcpy(&tmp, i, sizeof(int), hipMemcpyDeviceToHost);
    
    return tmp;
}

void printEdges(std::vector<edge> edges){
    for(edge e : edges){
        printf("src: %d, dst: %d, weight: %d\n", e.src, e.dest, e.weight);
    }
}

int getNumVertices(std::vector<edge> edges){
    int max = -1;

    // vertices will be numbered starting from 0 or 1
    bool verticesStartFromZero = false;

    for(edge e : edges){
        int tmp = std::max(e.src, e.dest);
        max = std::max(max, tmp);

        if(e.src == 0){
            verticesStartFromZero = true;
        }
    }

    return verticesStartFromZero ? max + 1 : max;
}

void writeAnswer(int *output, int len){
    FILE *fp = fopen("output.txt", "w");
    for(int i = 0; i < len; i++){
        fprintf(fp, "%d:\t%d\n", i, output[i]);
    }
    fclose(fp);
}

__global__ void cudaInitIntArray(int *a, int len, int val){
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (len % totalWarps == 0) ? len / totalWarps : len / totalWarps + 1;
    int beg = load * warpId;
    int end = (len < beg + load) ? len : beg + load;
    beg = beg + laneId;

    for(int i = beg; i < end; i += 32){
        a[i] = val;
    }
}

struct vertex{
  int start;
  unsigned int len;
  int successor;
};

/****** END UTIL METHODS ******/

bool done(vertex *v, int vlen){
    vertex tmp;
    hipMemcpy(&tmp, v, sizeof(vertex), hipMemcpyDeviceToHost);
    int successor = tmp.successor;
    for(int i = 1; i < vlen; i++){
        hipMemcpy(&tmp, &v[i], sizeof(vertex), hipMemcpyDeviceToHost);
        if(successor != tmp.successor){
            return false;
        }
    }

    return true;
}

__global__ void
findMins(vertex *v, edge *e, int *inMst, int vlen, int elen){
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (vlen % totalWarps == 0) ? vlen / totalWarps : vlen / totalWarps + 1;
    int beg = load * warpId;
    int end = (vlen < beg + load) ? vlen : beg + load;
    beg = beg + laneId;

    for(int i = beg; i < end; i += 32){
        vertex cur = v[i];
        int min = INT_MAX;
        int minIndex = elen;
        int minv = INT_MAX;
        for(int j = cur.start; j < (cur.start + cur.len); j++){
            int w = e[j].weight;
            int tmpMin = min; int tmpInd = minIndex;
            int dest = e[j].dest;
            min = ( ((min < w) || ((min == w) && (minv < dest))) * min) + ( (((w == min) && (dest < minv)) || (w < min)) * w);
            minIndex = ( ((tmpMin < w) || ((tmpMin == w) && (minv < dest))) * minIndex) + ( (((w == tmpMin) && (dest < minv)) || (w < tmpMin)) * j);
            minv = ( ((tmpMin < w) || ((tmpMin == w) && (minv < dest))) * minv) + ( (((w == tmpMin) && (dest < minv)) || (w < tmpMin)) * dest);
            //printf("minv: %d, cursuc: %d\n", minv, cur.successor);

            //minIndex = ((tmpMin <= w) * minIndex) + (((w < tmpMin) && (j < minIndex)) * j);
            //printf("min: %d\tnew: %d\t%d\n", tmpMin, w, (int)(((w == tmpMin) && (dest < minv)) || (w < tmpMin)));
        }
        inMst[minIndex] = 1;
        //printf("min index: %d", minIndex);
        e[minIndex].weight = INT_MAX;
        v[i].successor = ((minv != INT_MAX) * minv) + ((minv == INT_MAX) * cur.successor);
        printf("cur: %d, suc: %d\n", i, v[i].successor);
    }
}

__global__ void
setSuccessors(vertex *v, int vlen){
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (vlen % totalWarps == 0) ? vlen / totalWarps : vlen / totalWarps + 1;
    int beg = load * warpId;
    int end = (vlen < beg + load) ? vlen : beg + load;
    beg = beg + laneId;

    for(int i = beg; i < end; i += 32){
        while(v[v[i].successor].successor != v[i].successor){
            v[i].successor = v[v[i].successor].successor;
        }
    }
}

__global__ void
fixSuccessors(vertex *v, int vlen){
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (vlen % totalWarps == 0) ? vlen / totalWarps : vlen / totalWarps + 1;
    int beg = load * warpId;
    int end = (vlen < beg + load) ? vlen : beg + load;
    beg = beg + laneId;

    for(int i = beg; i < end; i += 32){
        vertex cur = v[i];
        bool shouldChange = (v[cur.successor].successor == i) && (i < cur.successor);
        printf("me: %d, suc: %d, sucsuc: %d\n", i, cur.successor, v[cur.successor].successor);
        v[i].successor = ((int)shouldChange * i) + (((int)!shouldChange) * cur.successor);
        //printf("cur: %d, suc: %d\n", i, cur.successor);
    }
}

void mst(std::vector<edge> * edgesPtr, int blockSize, int blockNum){
    setTime();
    
    // get edge list
    std::vector<edge> edgeVector = *edgesPtr;
    std::sort(edgeVector.begin(), edgeVector.end(), edgeSrcComparator);
    edge *edges = edgeVector.data();
    int elen = edgeVector.size();

    // get vertex list
    vertex *vertices;
    int vlen = getNumVertices(edgeVector);
    vertices = (vertex*)malloc(vlen * sizeof(vertex));
    for(int i = 0; i < vlen; i++){
        vertices[i].start = INT_MAX;
        vertices[i].successor = i;
    }
    int prevSrc = -1;
    int curVertex = 0;
    int start = 0;
    int len = 0;
    for(int i = 0; i < edgeVector.size(); i++){
        edge e = edgeVector[i];
      if(prevSrc == -1){
        curVertex = e.src;
        prevSrc = e.src;
      }

      if(prevSrc != e.src){
        vertices[curVertex].start = start;
        //printf("cur: %d, start: %d", curVertex, vertices[curVertex].start);
        vertices[curVertex].len = len;
        
        start += len;
        curVertex = e.src;
        prevSrc = e.src;
        len = 0;
      }

      len++;
    }
    vertices[curVertex].start = start;
    vertices[curVertex].len = len;

    for(int i = 0; i < vlen; i++){
        printf("v%d: start = %d len = %d %d\n", i, vertices[i].start, vertices[i].len, vlen);
    }
    for(int i = 0; i < elen; i++){
        printf("e%d: src = %d dest = %d weight = %d\n", i, edges[i].src, edges[i].dest, edges[i].weight);
    }

    int *inMst;
    hipMalloc((void**)&inMst, sizeof(int) * elen);
    cudaInitIntArray<<<blockNum, blockSize>>>(inMst, elen, 0);

    edge *e; 
    hipMalloc((void**)&e, sizeof(edge) * elen);
    hipMemcpy(e, edgeVector.data(), elen * sizeof(edge), hipMemcpyHostToDevice);

    vertex *v;
    hipMalloc((void**)&v, sizeof(vertex) * vlen);
    hipMemcpy(v, vertices, vlen * sizeof(vertex), hipMemcpyHostToDevice);

    int stop = 0;
    while(!done(v, vlen)){
        findMins<<<blockSize, blockNum>>>(v, e, inMst, vlen, elen);

        /*for(int i = 0; i < elen; i++){
            printf("HERE %d:\t%d\n", i, readCudaInt(&inMst[i]));
        }*/
        for(int i = 0; i < vlen; i++){
            vertex tmp;
            hipMemcpy(&tmp, &v[i], sizeof(vertex), hipMemcpyDeviceToHost);
            printf("HERE %d:\t%d\n", i, tmp.successor);
        }
        fixSuccessors<<<blockSize, blockNum>>>(v, vlen);
        hipDeviceSynchronize();
        setSuccessors<<<blockSize, blockNum>>>(v, vlen);
        for(int i = 0; i < vlen; i++){
            vertex tmp;
            hipMemcpy(&tmp, &v[i], sizeof(vertex), hipMemcpyDeviceToHost);
            printf("HERE %d:\t%d\n", i, tmp.successor);
        }

        if(stop >= 2){
            break;
        }
        stop++;

        for(int i = 0; i < elen; i++){
            if(readCudaInt(&inMst[i]) == 1){
                edge tmp = edges[i];
                printf("IN MST: %d\t%d\n", tmp.src, tmp.dest);
            }
        }
    }

    hipDeviceProp_t props; hipGetDeviceProperties(&props, 0);
    printf("The total computation kernel time on GPU %s is %f milli-seconds\n", props.name, getTime());
}
